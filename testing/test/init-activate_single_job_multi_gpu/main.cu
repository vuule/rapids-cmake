

#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>


int main() {
  int nDevices = 0;
  hipGetDeviceCount(&nDevices);

  //We should have anywhere from 1 to 2 devices allocated
  std::cout << "Seeing " << nDevices << " GPU devices" << std::endl;

  if (nDevices == 0 || nDevices > 2) {
    return 1;
  }
  return 0;
}
