

#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>


int main() {

  // Very we only have a single GPU visible to us
  int nDevices = 0;
  hipGetDeviceCount(&nDevices);

  std::cout << "Seeing " << nDevices << " GPU devices" << std::endl;

  if (nDevices == 0 || nDevices > 3) {
    return 1;
  }
  return 0;
}
