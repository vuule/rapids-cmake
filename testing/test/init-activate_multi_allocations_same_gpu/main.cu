

#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>


int main() {

  // Very we only have a single GPU visible to us
  int nDevices = 0;
  hipGetDeviceCount(&nDevices);

  if (nDevices == 0) {
    return 1;
  }
  std::cout << "Seeing at least a single GPU" << std::endl;
  return 0;
}
