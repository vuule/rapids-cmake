/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <string>
#include <vector>

struct version {
  int major = 1;
  int minor = 0;
};

struct gpu {
  gpu(int i) : id{i} {};
  gpu(int i, const hipDeviceProp_t& prop) : id{i}, memory{prop.totalGlobalMem}, slots{100} {}
  int id        = 0;
  size_t memory = 0;
  int slots     = 0;
};

struct local {
  local()
  {
    int nDevices = 0;
    hipGetDeviceCount(&nDevices);
    if (nDevices == 0) {
      gpus.emplace_back(0);
    } else {
      for (int i = 0; i < nDevices; ++i) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        gpus.emplace_back(i, prop);
      }
    }
  }
  std::vector<gpu> gpus;
};

// A hard-coded JSON printer, but it keeps the job done
void to_json(std::ostream& buffer, version const& v)
{
  buffer << "\"version\":{\"major\":" << v.major << ", \"minor\": " << v.minor << "}";
}
void to_json(std::ostream& buffer, gpu const& g)
{
  buffer << "\t\t{\"id\":\"" << g.id << "\", \"slots\": " << g.slots << "}";
}
void to_json(std::ostream& buffer, local const& l)
{
  buffer << "\"local\": [ {\n";
  buffer << "\t\"gpus\": [\n";
  for (int i = 0; i < l.gpus.size(); ++i) {
    to_json(buffer, l.gpus[i]);
    if (i != (l.gpus.size() - 1)) { buffer << ","; }
    buffer << "\n";
  }
  buffer << "\t]\n";
  buffer << "} ]\n";
}

int main(int, char**)
{
  version v{1, 0};
  local l;

  std::cout << "{\n";
  to_json(std::cout, v);
  std::cout << ",\n";
  to_json(std::cout, l);
  std::cout << "}" << std::endl;
  return 0;
}